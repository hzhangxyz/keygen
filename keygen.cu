#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ char key[] = "$1&1234-1234-123456";

__device__ int f(int n, int byte, int c) {
  for (int bitIndex = 0; bitIndex <= 7; bitIndex++) {
    int bit = (byte >> bitIndex) & 1;
    if (bit + ((n - bit) & ~1) == n) {
      n = (n - bit) >> 1;
    } else {
      n = ((c - bit) ^ n) >> 1;
    }
  }
  return n;
}

__global__ void keygen(char* mathId, int hash_base, char* res) {
  res += 16*(blockIdx.x*blockDim.x+threadIdx.x);
  int hash = hash_base + blockIdx.x*blockDim.x + threadIdx.x;

  for(int byteIndex = 18; byteIndex >= 0; byteIndex--){
    hash = f(hash, (int)key[byteIndex], 0x105C3);
  }
  for(int byteIndex = 15; byteIndex >= 0; byteIndex--){
    hash = f(hash, (int)mathId[byteIndex], 0x105C3);
  }

  int n1 = 0;
  while (f(f(hash, n1 & 0xFF, 0x105C3), n1 >> 8, 0x105C3) != 0xA5B6) {
    ++n1;
  }

  n1 = floor(((n1 + 0x72FA) & 0xFFFF) * 99999.0 / 0xFFFF);
  int temp = n1/1000*1000 + n1%100*10 + n1%1000/100;
  temp = ceil((temp/99999.0)*0xFFFF);
  temp = f(f(0, temp & 0xFF, 0x1064B), temp >> 8, 0x1064B);

  for(int byteIndex = 18; byteIndex >= 0; byteIndex--){
    temp = f(temp, (int)key[byteIndex], 0x1064B);
  }
  for(int byteIndex = 15; byteIndex >= 0; byteIndex--){
    temp = f(temp, (int)mathId[byteIndex], 0x1064B);
  }

  int n2 = 0;
  while (f(f(temp, n2 & 0xFF, 0x1064B), n2 >> 8, 0x1064B) != 0xA5B6) {
    ++n2;
  }

  n2 = floor((n2 & 0xFFFF) * 99999.0 / 0xFFFF);

  res[10] = n1 % 10 + 48;
  res[1] = (n1/=10) % 10 + 48;
  res[6] = (n1/=10) % 10 + 48;
  res[2] = (n1/=10) % 10 + 48;
  res[3] = (n1/=10) % 10 + 48;
  res[5] = n2 % 10 + 48;
  res[0] = (n2/=10) % 10 + 48;
  res[9] = (n2/=10) % 10 + 48;
  res[11] =(n2/=10) % 10 + 48;
  res[7] = (n2/=10) % 10 + 48;
  res[4] = '-';
  res[8] = '-';
  res[12] = ':';
  res[13] = ':';
  res[14] = '1';
  res[15] = 0;
}

#if !defined Thread_Num
#define Thread_Num 1024
#endif

int main(int argc, char** argv){
  int hashStart = 0x0;
  int hashEnd   = 0x10000;

  if(argc==4){
    sscanf(argv[2],"%x",&hashStart);
    sscanf(argv[3],"%x",&hashEnd);
  }
  if(argc==3){
    sscanf(argv[2],"%x",&hashStart);
    hashEnd = hashStart + 1;
  }

  int Total_Number = hashEnd - hashStart;
  int Block_Num = (Total_Number + Thread_Num - 1)/ Thread_Num;

  char* math_id = NULL;
  size_t math_id_size = strlen(argv[1])*sizeof(char);
  hipMalloc((void**)&math_id, math_id_size);
  hipMemcpy(math_id, argv[1], math_id_size, hipMemcpyHostToDevice);

  char h_res[16*Block_Num*Thread_Num];
  char* d_res = NULL;
  size_t res_size = 16*Block_Num*Thread_Num*sizeof(char);
  hipMalloc((void**)&d_res, res_size);

  printf("Hash      MathId             Key           Password\n");

  keygen<<<Block_Num,Thread_Num>>>(math_id,hashStart,d_res);
  hipMemcpy(h_res, d_res, 16*Total_Number*sizeof(char), hipMemcpyDeviceToHost);
  for(int hash_del = 0; hash_del<Total_Number; hash_del++){
    printf("%04X %s 1234-1234-123456 %s\n",
            hashStart+hash_del,
            argv[1],
            h_res+16*hash_del);
  }

  return 0;
}
